#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "includes/per_ray_data_gpu.h"

using namespace optix;

rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(optix::float3, front_hit_point, attribute front_hit_point,);

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );


RT_PROGRAM void any_hit()
{
    // this material is opaque, so it fully attenuates all shadow rays
    prd_shadow.visible = false;
    rtTerminateRay();
}

RT_PROGRAM void closest_hit()
{
    prd_radiance.radiance = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal))*0.5f + 0.5f;
}
