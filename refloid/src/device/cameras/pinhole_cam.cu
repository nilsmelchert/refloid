#include "hip/hip_runtime.h"
#include "pinhole_cam.h"

// Camera Parameters
rtDeclareVariable(unsigned int, width, ,);
rtDeclareVariable(unsigned int, height, ,);
rtDeclareVariable(optix::Matrix4x4, K, ,);
rtDeclareVariable(optix::Matrix4x4, K_inv, ,);
rtDeclareVariable(optix::Matrix4x4, Rt, ,);
rtDeclareVariable(optix::Matrix4x4, Rt_inv, ,);
rtBuffer<float>distBuff;
rtBuffer<float>undistBuff;

rtDeclareVariable(float, scene_epsilon, ,);

// Buffers for image data
rtBuffer<uchar4, 2> sysOutputBuffer; // RGB32F
rtBuffer<float4, 2> sysAccumBuffer; // RGB32F

// Top object which was declared as m_root_group in RT_scene at host side
rtDeclareVariable(rtObject, sysTopObject, ,);
rtDeclareVariable(unsigned int, frame, ,);
rtDeclareVariable(uint2, launch_index, rtLaunchIndex,);

RT_PROGRAM void camera()
{
    optix::size_t2 screen = sysOutputBuffer.size();

    // Convert pixel so that it follows the OpenCV convention
    uint2 launch_index_cv = launch_index;
    launch_index_cv.y = (height - 1) - launch_index_cv.y;

    // Provides a random number between -veryhighnumber and +veryhighnumber
    unsigned int seed = tea<16>(screen.x * launch_index_cv.y + launch_index_cv.x, frame);
    //Subpixel jitter: send the ray through a different position inside the pixel each time,
    // to provide antialiasing.
    // Random number generator (the value is between 0 and 1
    float2 subpixel_jitter = frame == 0 ? make_float2(0.0f) : make_float2(rng(seed) - 0.5f, rng(seed) - 0.5f);

    // d is pixel for a casted ray
    float2 d = (make_float2(launch_index_cv) + subpixel_jitter);

    // calculate distortion from OpenCV distortion parameters
    distort_pixels(d);
    // calculate ray direction from OpenCV pinhole camera model parameters
    optix::float3 ray_direction = calculate_ray_direction(d);

    // Apply extrinsic transformation (position of the camera with respect to the world coordinate system)
    ray_direction = optix::make_matrix3x3(Rt) * ray_direction;
    float3 ray_origin = make_float3(Rt[3], Rt[7], Rt[11]);

    PerRayData_radiance prd = init_per_ray_data();
    optix::Ray ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(sysTopObject, ray, prd);

    // NaN values will never go away. Filter them out before they can arrive in the output buffer.
    // This only has an effect if the debug coloring above is off!
    if (!(isnan(prd.radiance.x) || isnan(prd.radiance.y) || isnan(prd.radiance.z)))
    {
        float4 acc_val = sysAccumBuffer[launch_index];
        if (frame > 0) {
            acc_val = lerp( acc_val, make_float4( prd.radiance, 0.f ), 1.0f / static_cast<float>( frame+1 ) );
        }else {
            acc_val = optix::make_float4(prd.radiance, 1.0f);
        }
        sysOutputBuffer[launch_index] = make_color( optix::make_float3(acc_val));
        sysAccumBuffer[launch_index] = acc_val;
    }
}

RT_FUNCTION void distort_pixels(float2 &uv)
{
    float X, Y, Z, r2, u_, v_;
    X = uv.x * K_inv[0] + uv.y * 0.0f + K_inv[2];
    Y = uv.x * 0.0f + uv.y * K_inv[5] + K_inv[6];
    Z = uv.x * 0.0f + uv.y * 0.0f + 1.0f;
    X /= Z; Y /= Z;
    //radial^2
    r2 = X*X + Y*Y;
    u_ = X * (1.0f + distBuff[0]*r2 + distBuff[1]*(r2*r2) + distBuff[4]*(r2*r2*r2)) + 2.0f*distBuff[2]*(X*Y) + distBuff[3]*(r2 + 2.0f*(X*X));
    v_ = Y * (1.0f + distBuff[0]*r2 + distBuff[1]*(r2*r2) + distBuff[4]*(r2*r2*r2)) + distBuff[2]*(r2 + 2.0f*(Y*Y)) + 2.0f*distBuff[3]*(X*Y);

    X = u_ * K[0] + v_ * 0.0f + K[2];
    Y = u_ * 0.0f + v_ * K[5] + K[6];
    Z = u_ * 0.0f + v_ * 0.0f + 1.0f;
    //de-homogenize
    uv.x = X / Z;
    uv.y = Y / Z;
}

RT_FUNCTION optix::float3 calculate_ray_direction(float2 &uv)
{
    return normalize(make_float3((1.0f / K[0]) * (uv.x - K[2]), (1.0f / K[5]) * (uv.y - K[6]), 1.0f));
}

RT_FUNCTION PerRayData_radiance init_per_ray_data()
{
    PerRayData_radiance prd;
    prd.depth = 0; // Initialize Bounces
//    prd.intensity = 0.0f; //Initialize Multireflection overall Intensity for Calculations
    // These represent the current shading state and will be set by the closest-hit or miss program
    // attenuation (<= 1) from surface interaction.
    prd.reflectance = make_float3(1.0f);
    // light from a light source or miss program
    prd.radiance = make_float3(0.0f);
    // next ray to be traced
    prd.origin = make_float3(0.0f);
    prd.direction = make_float3(0.0f);
    return prd ;
}