#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_cuda.h>
#include <valarray>
#include <optix_world.h>

#include "includes/helpers_gpu.h"
#include "includes/per_ray_data_gpu.h"
#include "includes/random_number_generators_gpu.h"

rtDeclareVariable(unsigned int, entry_point_id, ,);
rtDeclareVariable(int, parCameraIndex,,);

// Camera Parameters
rtDeclareVariable(unsigned int, width, ,);
rtDeclareVariable(unsigned int, height, ,);
rtDeclareVariable(optix::Matrix4x4, K, ,);
rtDeclareVariable(optix::Matrix4x4, K_inv, ,);
rtDeclareVariable(optix::Matrix4x4, Rt, ,);
rtDeclareVariable(optix::Matrix4x4, Rt_inv, ,);
rtBuffer<float>distBuff;
rtBuffer<float>undistBuff;


rtDeclareVariable(float, scene_epsilon, ,);
rtDeclareVariable(float3, cutoff_color, ,);
rtDeclareVariable(int, max_depth, ,);


rtBuffer<uchar4, 2> sysOutputBuffer; // RGB32F
rtBuffer<float4, 2> sysAccumBuffer; // RGB32F

rtDeclareVariable(rtObject, sysTopObject, ,);
rtDeclareVariable(unsigned int, frame, ,);
rtDeclareVariable(uint2, launch_index, rtLaunchIndex,);

rtDeclareVariable(unsigned int, radiance_ray_type, ,);

RT_FUNCTION void distort_pixels(float2 &uv);

RT_FUNCTION optix::float3 calculate_ray_direction(float2 &uv);

RT_PROGRAM void camera() {
    optix::size_t2 screen = sysOutputBuffer.size();

    // Convert pixel so that it follows the OpenCV convention
    uint2 launch_index_cv = launch_index;
    launch_index_cv.y = (height - 1) - launch_index_cv.y;

    // Provides a random number between -veryhighnumber and +veryhighnumber
    unsigned int seed = tea<16>(screen.x * launch_index_cv.y + launch_index_cv.x, frame);
    //Subpixel jitter: send the ray through a different position inside the pixel each time,
    // to provide antialiasing.
    // Random number generator (the value is between 0 and 1
    float2 subpixel_jitter = frame == 0 ? make_float2(0.0f) : make_float2(rng(seed) - 0.5f, rng(seed) - 0.5f);

    // d is pixel for a casted ray
    float2 d = (make_float2(launch_index_cv) + subpixel_jitter);// / make_float2(screen) * 2.f - 1.f;

    distort_pixels(d);

    optix::float3 ray_direction = calculate_ray_direction(d);

    // Apply extrinsic transformation
    ray_direction = optix::make_matrix3x3(Rt) * ray_direction;
    float3 ray_origin = make_float3(Rt[3], Rt[7], Rt[11]);

    PerRayData_radiance prd;
    prd.depth = 0; // Initialize Bounces
    prd.seed = seed; // Random generated Number (Sample) for spatial antialiasing
    prd.intensity = 0.0f; //Initialize Multireflection overall Intensity for Calculations

    // These represent the current shading state and will be set by the closest-hit or miss program
    // attenuation (<= 1) from surface interaction.
    prd.reflectance = make_float3(1.0f);
    // light from a light source or miss program
    prd.radiance = make_float3(0.0f);
    // next ray to be traced
    prd.origin = make_float3(0.0f);
    prd.direction = make_float3(0.0f);
    optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(sysTopObject, ray, prd);

    // NaN values will never go away. Filter them out before they can arrive in the output buffer.
    // This only has an effect if the debug coloring above is off!
    if (!(isnan(prd.radiance.x) || isnan(prd.radiance.y) || isnan(prd.radiance.z)))
    {
        float4 acc_val = sysAccumBuffer[launch_index];
        if (frame > 0) {
            acc_val = lerp( acc_val, make_float4( prd.radiance, 0.f ), 1.0f / static_cast<float>( frame+1 ) );
        }else {
            acc_val = optix::make_float4(prd.radiance, 1.0f);
        }
        sysOutputBuffer[launch_index] = make_color( optix::make_float3(acc_val));

        sysAccumBuffer[launch_index] = acc_val;
        // Change here for a gamma corrected RGB picture
    }
}

RT_FUNCTION void distort_pixels(float2 &uv)
{
    float X, Y, Z, r2, u_, v_;
    X = uv.x * K_inv[0] + uv.y * 0.0f + K_inv[2];
    Y = uv.x * 0.0f + uv.y * K_inv[5] + K_inv[6];
    Z = uv.x * 0.0f + uv.y * 0.0f + 1.0f;
    X /= Z; Y /= Z;
    //radial^2
    r2 = X*X + Y*Y;
    u_ = X * (1.0f + distBuff[0]*r2 + distBuff[1]*(r2*r2) + distBuff[4]*(r2*r2*r2)) + 2.0f*distBuff[2]*(X*Y) + distBuff[3]*(r2 + 2.0f*(X*X));
    v_ = Y * (1.0f + distBuff[0]*r2 + distBuff[1]*(r2*r2) + distBuff[4]*(r2*r2*r2)) + distBuff[2]*(r2 + 2.0f*(Y*Y)) + 2.0f*distBuff[3]*(X*Y);

    X = u_ * K[0] + v_ * 0.0f + K[2];
    Y = u_ * 0.0f + v_ * K[5] + K[6];
    Z = u_ * 0.0f + v_ * 0.0f + 1.0f;
    //de-homogenize
    uv.x = X / Z;
    uv.y = Y / Z;
}

RT_FUNCTION optix::float3 calculate_ray_direction(float2 &uv)
{
    return normalize(make_float3((1.0f / K[0]) * (uv.x - K[2]), (1.0f / K[5]) * (uv.y - K[6]), 1.0f));
}
