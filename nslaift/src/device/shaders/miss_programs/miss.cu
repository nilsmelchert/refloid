#include "hip/hip_runtime.h"
/*
   @file    miss.cu
  @author   Pascal Kern   (kern@imr.uni-hannover.de)
  @since    2018 / 1 / 8
  @brief    Coloring Background with Miss Color set in main
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_cuda.h>

#include "includes/per_ray_data_gpu.h"

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(float3, miss_color, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void miss_environment_constant()
{
    prd_radiance.is_miss = true;
    prd_radiance.radiance = miss_color; // Background Color set in Host (black)
}
