#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_cuda.h>
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include <internal/optix_defines.h>

#include "includes/per_ray_data_gpu.h"
#include "includes/helpers_gpu.h"

using namespace optix;

// TODO: need to be implemented
//rtDeclareVariable(int,               max_depth, , );
//rtBuffer<BasicLight>                 lights;
//rtDeclareVariable(float3,            ambient_light_color, , );
//rtDeclareVariable(float,             scene_epsilon, , );
//rtDeclareVariable(rtObject,          top_object, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable(float, intersectionDist, rtIntersectionDistance, );

rtDeclareVariable(float, scene_epsilon, ,);

RT_PROGRAM void any_hit()
{
    // this material is opaque, so it fully attenuates all shadow rays
    rtTerminateRay();
    prd_shadow.visible = false;
}

RT_PROGRAM void closest_hit()
{
    float3 front_hit_point = ray.origin + intersectionDist * ray.direction;

    // Transform normals to world coordinates
    const float3 world_shading_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal) );
    const float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

    //Calculating the face forward normal N
    const float3 N = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    // Vector to camera
    const float3 wo = -ray.direction;

    // Transform front hit point to world coordinates
    const float3 fhp_world = rtTransformPoint(RT_OBJECT_TO_WORLD, front_hit_point);

    prd_radiance.origin = fhp_world;

    //Intialize radiance
    float3 radiance = make_float3(0.0f);

    // Initialize intensity
//    float intensity = 0.0f;


//    for (int i; i<num_lights; i++) {
//        //Dummy temporary light
         float3 light_direction = -ray.direction;
//      TODO: LICHTER HIER
//    }

    PerRayData_shadow prdShadow;
    prdShadow.visible = true; // Initialize for miss.
    // Note that the sysSceneEpsilon is applied on both sides of the shadow ray [t_min, t_max] interval
    // to prevent self intersections with the actual light_definition geometry in the scene!
//    optix::Ray shadow_ray = optix::make_Ray(prd_radiance.origin, normalize(light_direction), SHADOW_RAY_TYPE,
//                                            scene_epsilon,
//                                            light_definition.distance - scene_epsilon);
//    rtTrace(top_object, shadow_ray, prdShadow); // Trace Shadow Ray



    // Shading with Phong-BRDF using the parameters Kd, Ka, Ks, Kr, phong_exp and ffnormal

//    float3 hit_point = ray.origin + t_hit * ray.direction;

}
