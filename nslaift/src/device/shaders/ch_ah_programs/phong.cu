#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include <internal/optix_defines.h>

#include "includes/per_ray_data_gpu.h"

using namespace optix;

// TODO: need to be implemented
//rtDeclareVariable(int,               max_depth, , );
//rtBuffer<BasicLight>                 lights;
//rtDeclareVariable(float3,            ambient_light_color, , );
//rtDeclareVariable(float,             scene_epsilon, , );
//rtDeclareVariable(rtObject,          top_object, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable(float, intersectionDist, rtIntersectionDistance, );

RT_PROGRAM void any_hit()
{
    // this material is opaque, so it fully attenuates all shadow rays
    rtTerminateRay();
    prd_shadow.visible = false;
}

RT_PROGRAM void closest_hit()
{
    float3 front_hit_point = ray.origin + intersectionDist * ray.direction;

    const float3 world_shading_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal) );
    const float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    //Calculating the face forward normal N
    const float3 N = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    const float3 wo = -ray.direction; // Vector to camera

    const float3 fhp_world = rtTransformPoint(RT_OBJECT_TO_WORLD, front_hit_point);

    prd_radiance.origin = fhp_world;

//    for (int i; i<num_lights; i++) {
//        //Do stuff here
//      LICHTER HIER
//    }



    // Shading with Phong-BRDF using the parameters Kd, Ka, Ks, Kr, phong_exp and ffnormal

//    float3 hit_point = ray.origin + t_hit * ray.direction;

}
