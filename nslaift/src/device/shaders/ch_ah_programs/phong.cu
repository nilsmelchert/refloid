#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_cuda.h>
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include <internal/optix_defines.h>

#include "includes/per_ray_data_gpu.h"
#include "includes/helpers_gpu.h"
#include "includes/app_config.h"
#include "includes/light_definition.h"

using namespace optix;

// Attributes
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

// Semantics
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, intersectionDist, rtIntersectionDistance, );

rtDeclareVariable(float, scene_epsilon, ,);
rtDeclareVariable(rtObject, sysTopObject, ,);
rtDeclareVariable(unsigned int, light_count, ,);

// BRDF specific variables (phong)
rtDeclareVariable(optix::float3, Kd, ,);
rtDeclareVariable(optix::float3, Ks, ,);
rtDeclareVariable(float, specular_exponent, ,);

rtBuffer<rtCallableProgramId<void(float3 const &point, const float2 sample,
                                  LightDefinition &lightDef)> > sysLightBuffer;

RT_PROGRAM void any_hit()
{
    // this material is opaque, so it fully attenuates all shadow rays
    prd_shadow.visible = false;
    rtTerminateRay();
}

RT_PROGRAM void closest_hit()
{
    // Calculate front hit point in object coordinates
    float3 fhp = ray.origin + intersectionDist * ray.direction;

    // Transform normals to world coordinates
    const float3 world_shading_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal) );
    const float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

    //Calculating the face forward normal N in world coordinates
    const float3 N = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    // Vector to camera
    const float3 wo = -ray.direction;

    // Transform front hit point to world coordinates
    const float3 fhp_world = rtTransformPoint(RT_OBJECT_TO_WORLD, fhp);

    prd_radiance.origin = fhp_world;

    LightDefinition light_def;
    for (int i=0; i<light_count; i++) {
//      TODO: LICHTER HIER. FOR NOW DUMMY:
        sysLightBuffer[i](fhp_world, make_float2(1.0f), light_def);


        PerRayData_shadow prdShadow;
        prdShadow.visible = true; // Initialize for miss.
        // Note that the sysSceneEpsilon is applied on both sides of the shadow ray [t_min, t_max] interval
        // to prevent self intersections with the actual light_definition geometry in the scene!
        optix::Ray shadow_ray = optix::make_Ray(prd_radiance.origin, light_def.wi, SHADOW_RAY_TYPE,
                                                scene_epsilon,
                                                light_def.distance - scene_epsilon);
        rtTrace(sysTopObject, shadow_ray, prdShadow); // Trace Shadow Ray

        if (prdShadow.visible) {
            const float cosAngIncidence = optix::clamp(optix::dot(N, light_def.wi), 0.0f, 1.0f);
            const float3 R = optix::normalize(2 * cosAngIncidence * N - light_def.wi);
            float phong_term = fmaxf(optix::dot(R, wo), 0.0f);
            phong_term = cosAngIncidence > DENOMINATOR_EPSILON ? phong_term : 0.0f;
            phong_term = powf(phong_term, specular_exponent);
            float3 f_phong_specular = make_float3(0.0f);
            if (cosAngIncidence > DENOMINATOR_EPSILON) { //Catch 0 division error
                f_phong_specular = optix::clamp(
                        (Ks * phong_term * M_PIf) / cosAngIncidence, make_float3(0.0f),
                        make_float3(1.0f)); //specular phong coefficient ks=material specific
            }
            const float3 f_phong_diffuse = Kd; //diffuse phong coefficient kd=material specific
            const float3 f_phong =
                    f_phong_diffuse + f_phong_specular;// Do the visibility check of the light_definition sample.

            prd_radiance.radiance +=
                    f_phong * light_def.emission /*light emission*/ * optix::dot(N, light_def.wi) * 1.0f /*solid angle*/;
        }
    }
}
