#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_cuda.h>
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include <internal/optix_defines.h>

#include "includes/per_ray_data_gpu.h"
#include "includes/helpers_gpu.h"
#include "includes/app_config.h"

using namespace optix;

// TODO: need to be implemented
//rtDeclareVariable(int,               max_depth, , );
//rtBuffer<BasicLight>                 lights;
//rtDeclareVariable(float3,            ambient_light_color, , );
//rtDeclareVariable(float,             scene_epsilon, , );
//rtDeclareVariable(rtObject,          top_object, , );

// Attributes
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

// Semantics
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, intersectionDist, rtIntersectionDistance, );

rtDeclareVariable(float, scene_epsilon, ,);
rtDeclareVariable(rtObject, sysTopObject, ,);

// BRDF specific variables (phong)
rtDeclareVariable(optix::float3, Kd, ,);
rtDeclareVariable(optix::float3, Ks, ,);
rtDeclareVariable(float, specular_exponent, ,);

RT_PROGRAM void any_hit()
{
    // this material is opaque, so it fully attenuates all shadow rays
    prd_shadow.visible = false;
    rtTerminateRay();
}

RT_PROGRAM void closest_hit()
{
    // Calculate front hit point in object coordinates
    float3 fhp = ray.origin + intersectionDist * ray.direction;

    // Transform normals to world coordinates
    const float3 world_shading_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal) );
    const float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

    //Calculating the face forward normal N in world coordinates
    const float3 N = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    // Vector to camera
    const float3 wo = -ray.direction;

    // Transform front hit point to world coordinates
    const float3 fhp_world = rtTransformPoint(RT_OBJECT_TO_WORLD, fhp);

    prd_radiance.origin = fhp_world;

/////////////////////////////////////////////////////
//    for (int i; i<num_lights; i++) {
//      TODO: LICHTER HIER. FOR NOW DUMMY:
        optix::float3 light_pos = make_float3(0.0f) - fhp_world;
        float light_distance = length(light_pos - fhp_world);
        optix::float3 light_wi = normalize(light_pos - fhp_world);
        float light_area = 1.0f;
//    }
////////////////////////////////////////////////////////////

    PerRayData_shadow prdShadow;
    prdShadow.visible = true; // Initialize for miss.
    // Note that the sysSceneEpsilon is applied on both sides of the shadow ray [t_min, t_max] interval
    // to prevent self intersections with the actual light_definition geometry in the scene!
    optix::Ray shadow_ray = optix::make_Ray(prd_radiance.origin, light_wi, SHADOW_RAY_TYPE,
                                            scene_epsilon,
                                            light_distance - scene_epsilon);
    rtTrace(sysTopObject, shadow_ray, prdShadow); // Trace Shadow Ray

    if (prdShadow.visible) {
        const float cosAngIncidence = optix::clamp(optix::dot(N, light_wi), 0.0f, 1.0f);
        const float3 R = optix::normalize(2 * cosAngIncidence * N - light_wi);
        float phong_term = fmaxf(optix::dot(R, wo), 0.0f);
        phong_term = cosAngIncidence > DENOMINATOR_EPSILON ? phong_term : 0.0f;
        phong_term = powf(phong_term, specular_exponent);
        float3 f_phong_specular = make_float3(0.0f);
        if (cosAngIncidence > DENOMINATOR_EPSILON) { //Catch 0 division error
            f_phong_specular = optix::clamp(
                    (Ks * phong_term * M_PIf) / cosAngIncidence, make_float3(0.0f),
                    make_float3(1.0f)); //specular phong coefficient ks=material specific
        }
        const float3 f_phong_diffuse = Kd; //diffuse phong coefficient kd=material specific
        const float3 f_phong = f_phong_diffuse + f_phong_specular;// Do the visibility check of the light_definition sample.

        prd_radiance.radiance += f_phong * 1.0f /*light emission*/ * optix::dot(N, light_wi) * 1.0f /*solid angle*/;
    }
}
