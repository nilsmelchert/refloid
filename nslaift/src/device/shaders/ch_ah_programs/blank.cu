#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "includes/per_ray_data_gpu.h"

using namespace optix;

rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );


RT_PROGRAM void any_hit()
{
    // this material is opaque, so it fully attenuates all shadow rays
    prd_shadow.visible = false;
    rtTerminateRay();
}

RT_PROGRAM void closest_hit()
{
    prd_radiance.radiance = make_float3(1.0f);
}
