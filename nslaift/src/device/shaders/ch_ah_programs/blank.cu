#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_world.h>
#include <optixu/optixu_math_namespace.h>
#include <internal/optix_defines.h>

#include "includes/per_ray_data_gpu.h"

using namespace optix;

rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

rtDeclareVariable(optix::float3, color, ,);

RT_PROGRAM void any_hit()
{
    // this material is opaque, so it fully attenuates all shadow rays
    prd_shadow.visible = false;
    rtTerminateRay();
}

RT_PROGRAM void closest_hit()
{
    prd_radiance.radiance = make_float3(1.0f);
}
