#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_cuda.h>
#include <valarray>
#include <optix_world.h>
//#include <zconf.h>
#include "helpers.h"
#include "per_ray_data.h"
#include "random_number_generators.h"
#include "app_config.h"

using namespace optix;

rtDeclareVariable(unsigned int, entry_point_id, ,);
rtDeclareVariable(int, parCameraIndex,,);

// Camera Parameters
rtDeclareVariable(unsigned int, width, ,);
rtDeclareVariable(unsigned int, height, ,);
rtDeclareVariable(optix::Matrix4x4, K, ,);
rtDeclareVariable(optix::Matrix4x4, K_inv, ,);
rtDeclareVariable(optix::Matrix4x4, Rt, ,);
rtDeclareVariable(optix::Matrix4x4, Rt_inv, ,);
rtBuffer<float>distBuff;
rtBuffer<float>undistBuff;


rtDeclareVariable(float, scene_epsilon, ,);
rtDeclareVariable(float3, cutoff_color, ,);
rtDeclareVariable(int, max_depth, ,);


rtBuffer<rtBufferId<uchar4, 2>, 1> output_id_buffer;
rtBuffer<rtBufferId<float4, 2>, 1> output_id_accum_buffer;
rtBuffer<rtBufferId<float, 2>, 1> output_id_intensity_buffer;

rtDeclareVariable(rtObject, top_object, ,);
rtDeclareVariable(unsigned int, frame, ,);
rtDeclareVariable(uint2, launch_index, rtLaunchIndex,);

rtDeclareVariable(unsigned int, radiance_ray_type, ,);

RT_FUNCTION void distort_pixels(float2 &uv)
{
    float X, Y, Z, r2, u_, v_;
    X = uv.x * K_inv[0] + uv.y * 0.0f + K_inv[2];
    Y = uv.x * 0.0f + uv.y * K_inv[5] + K_inv[6];
    Z = uv.x * 0.0f + uv.y * 0.0f + 1.0f;
    X /= Z; Y /= Z;
    //radial^2
    r2 = X*X + Y*Y;
    u_ = X * (1.0f + distBuff[0]*r2 + distBuff[1]*(r2*r2) + distBuff[4]*(r2*r2*r2)) + 2.0f*distBuff[2]*(X*Y) + distBuff[3]*(r2 + 2.0f*(X*X));
    v_ = Y * (1.0f + distBuff[0]*r2 + distBuff[1]*(r2*r2) + distBuff[4]*(r2*r2*r2)) + distBuff[2]*(r2 + 2.0f*(Y*Y)) + 2.0f*distBuff[3]*(X*Y);

    X = u_ * K[0] + v_ * 0.0f + K[2];
    Y = u_ * 0.0f + v_ * K[5] + K[6];
    Z = u_ * 0.0f + v_ * 0.0f + 1.0f;
    //de-homogenize
    uv.x = X / Z;
    uv.y = Y / Z;
}

RT_PROGRAM void camera() {

    optix::size_t2 screen = output_id_buffer[entry_point_id].size();

    uint2 launch_index_cv = launch_index;

    launch_index_cv.y = (height - 1) - launch_index_cv.y;
    // Provides a random number between -veryhighnumber and +veryhighnumber
    unsigned int seed = tea<16>(screen.x * launch_index_cv.y + launch_index_cv.x, frame);
    //Subpixel jitter: send the ray through a different position inside the pixel each time,
    // to provide antialiasing.
    // Random number generator (the value is between 0 and 1
    float2 subpixel_jitter = frame == 0 ? make_float2(0.0f) : make_float2(rng(seed) - 0.5f, rng(seed) - 0.5f);

    // d is pixel for a casted ray
    float2 d = (make_float2(launch_index_cv) + subpixel_jitter);// / make_float2(screen) * 2.f - 1.f;

    distort_pixels(d);

    float3 ray_direction = normalize(
            make_float3((1.0f / K[0]) * (d.x - K[2]), (1.0f / K[5]) * (d.y - K[6]),
                        1.0f));

    // Apply extrinsic transformation
    ray_direction = optix::make_matrix3x3(Rt) * ray_direction;
    float3 ray_origin = make_float3(Rt[3], Rt[7], Rt[11]);

    PerRayData_radiance prd;
    prd.depth = 0; // Initialize Bounces
    prd.seed = seed; // Random generated Number (Sample) for spatial antialiasing
    prd.intensity = 0.0f; //Initialize Multireflection overall Intensity for Calculations

    // These represent the current shading state and will be set by the closest-hit or miss program
    // attenuation (<= 1) from surface interaction.
    prd.reflectance = make_float3(1.0f);
    // light from a light source or miss program
    prd.radiance = make_float3(0.0f);
    // next ray to be traced
    prd.origin = make_float3(0.0f);
    prd.direction = make_float3(0.0f);
    optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(top_object, ray, prd);
    // NaN values will never go away. Filter them out before they can arrive in the output buffer.
    // This only has an effect if the debug coloring above is off!
    if (!(isnan(prd.radiance.x) || isnan(prd.radiance.y) || isnan(prd.radiance.z)))
    {
        float4 acc_val = output_id_accum_buffer[entry_point_id][launch_index];
        if (frame > 0) {
#if Visualization_Transparency_OFF
            acc_val = lerp(acc_val, make_float4(prd.radiance, 0.f), 1.0f / static_cast<float>( frame + 1 ));
#endif
        }else {
            acc_val = make_float4(prd.radiance, 0.f);
        }

        output_id_buffer[entry_point_id][launch_index] = make_color(make_float3(acc_val));
        output_id_accum_buffer[entry_point_id][launch_index] = acc_val;
        if (frame == 0)
        {
            output_id_intensity_buffer[entry_point_id][launch_index] = 0.0f;
        }
        else if(prd.visible && prd.radiance.x >0.001f && prd.radiance.y > 0.001f && prd.radiance.z > 0.001f){
            output_id_intensity_buffer[entry_point_id][launch_index] = prd.intensity;
        }
        // Change here for a gamma corrected RGB picture
    }
}
