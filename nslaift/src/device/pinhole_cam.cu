#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_cuda.h>
#include <valarray>
#include <optix_world.h>
//#include <zconf.h>
#include "helpers.h"
#include "per_ray_data.h"
#include "random_number_generators.h"
#include "app_config.h"

using namespace optix;

rtDeclareVariable(unsigned int, entry_point_id, ,);
rtDeclareVariable(int, parCameraIndex,,);

rtDeclareVariable(float, scene_epsilon, ,);
rtDeclareVariable(float3, cutoff_color, ,);
rtDeclareVariable(int, max_depth, ,);


rtBuffer<rtBufferId<uchar4, 2>, 1> output_id_buffer;
rtBuffer<rtBufferId<float4, 2>, 1> output_id_accum_buffer;
rtBuffer<rtBufferId<float, 2>, 1> output_id_intensity_buffer;

rtDeclareVariable(rtObject, top_object, ,);
rtDeclareVariable(unsigned int, frame, ,);
rtDeclareVariable(uint2, launch_index, rtLaunchIndex,);

rtDeclareVariable(unsigned int, radiance_ray_type, ,);

RT_FUNCTION void distort_pixels(float2 &uv, camera_information &camera_parameters)
{
    float X, Y, Z, r2, u_, v_;
    X = uv.x * camera_parameters.K_inv[0] + uv.y * 0.0f + camera_parameters.K_inv[2];
    Y = uv.x * 0.0f + uv.y * camera_parameters.K_inv[5] + camera_parameters.K_inv[6];
    Z = uv.x * 0.0f + uv.y * 0.0f + 1.0f;
    X /= Z; Y /= Z;
    //radial^2
    r2 = X*X + Y*Y;
    u_ = X * (1.0f + camera_parameters.distortion[0]*r2 + camera_parameters.distortion[1]*(r2*r2) + camera_parameters.distortion[4]*(r2*r2*r2)) + 2.0f*camera_parameters.distortion[2]*(X*Y) + camera_parameters.distortion[3]*(r2 + 2.0f*(X*X));
    v_ = Y * (1.0f + camera_parameters.distortion[0]*r2 + camera_parameters.distortion[1]*(r2*r2) + camera_parameters.distortion[4]*(r2*r2*r2)) + camera_parameters.distortion[2]*(r2 + 2.0f*(Y*Y)) + 2.0f*camera_parameters.distortion[3]*(X*Y);

    X = u_ * camera_parameters.K[0] + v_ * 0.0f + camera_parameters.K[2];
    Y = u_ * 0.0f + v_ * camera_parameters.K[5] + camera_parameters.K[6];
    Z = u_ * 0.0f + v_ * 0.0f + 1.0f;
    //de-homogenize
    uv.x = X / Z;
    uv.y = Y / Z;
}

RT_PROGRAM void camera() {

    optix::size_t2 screen = output_id_buffer[entry_point_id].size();

    uint2 launch_index_cv = launch_index;
    camera_information camera_parameters = sysCameraParameters[parCameraIndex];

    launch_index_cv.y = (camera_parameters.height - 1) - launch_index_cv.y;
    // Provides a random number between -veryhighnumber and +veryhighnumber
    unsigned int seed = tea<16>(screen.x * launch_index_cv.y + launch_index_cv.x, frame);
    //Subpixel jitter: send the ray through a different position inside the pixel each time,
    // to provide antialiasing.
    // Random number generator (the value is between 0 and 1
    float2 subpixel_jitter = frame == 0 ? make_float2(0.0f) : make_float2(rng(seed) - 0.5f, rng(seed) - 0.5f);

    // d is pixel for a casted ray
    float2 d = (make_float2(launch_index_cv) + subpixel_jitter);// / make_float2(screen) * 2.f - 1.f;

    distort_pixels(d, camera_parameters);

    float3 ray_direction = normalize(
            make_float3((1.0f / camera_parameters.K[0]) * (d.x - camera_parameters.K[2]), (1.0f / camera_parameters.K[5]) * (d.y - camera_parameters.K[6]),
                        1.0f));

    // Apply extrinsic transformation
    ray_direction = optix::make_matrix3x3(camera_parameters.Rt) * ray_direction;
    float3 ray_origin = make_float3(camera_parameters.Rt[3], camera_parameters.Rt[7], camera_parameters.Rt[11]);

    PerRayData_radiance prd;
    prd.depth = 0; // Initialize Bounces
    prd.seed = seed; // Random generated Number (Sample) for spatial antialiasing
    prd.intensity = 0.0f; //Initialize Multireflection overall Intensity for Calculations

    // These represent the current shading state and will be set by the closest-hit or miss program
    // attenuation (<= 1) from surface interaction.
    prd.reflectance = make_float3(1.0f);
    // light from a light source or miss program
    prd.radiance = make_float3(0.0f);
    // next ray to be traced
    prd.origin = make_float3(0.0f);
    prd.direction = make_float3(0.0f);
    optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
    rtTrace(top_object, ray, prd);
#if USE_DEBUG_EXCEPTIONS
    // DAR DEBUG Highlight numerical errors.
    if (isnan(prd.radiance.x) || isnan(prd.radiance.y) || isnan(prd.radiance.z))
      {
        prd.radiance = make_float3(1000000.0f, 0.0f, 0.0f); // super red
      }
      else if (isinf(prd.radiance.x) || isinf(prd.radiance.y) || isinf(prd.radiance.z))
      {
        prd.radiance = make_float3(0.0f, 1000000.0f, 0.0f); // super green
      }
      else if (prd.radiance.x < 0.0f || prd.radiance.y < 0.0f || prd.radiance.z < 0.0f)
      {
        prd.radiance = make_float3(0.0f, 0.0f, 1000000.0f); // super blue
      }
#else
    // NaN values will never go away. Filter them out before they can arrive in the output buffer.
    // This only has an effect if the debug coloring above is off!
    if (!(isnan(prd.radiance.x) || isnan(prd.radiance.y) || isnan(prd.radiance.z)))
#endif
    {
        float4 acc_val = output_id_accum_buffer[entry_point_id][launch_index];
        if (frame > 0) {
#if Visualization_Transparency_OFF
            acc_val = lerp(acc_val, make_float4(prd.radiance, 0.f), 1.0f / static_cast<float>( frame + 1 ));
#endif
#if Multireflection_Visualization
        if (prd.depth > 0 && prd.visible && prd.radiance.x > 0.001f && prd.radiance.y > 0.001f && prd.radiance.z > 0.001f)
        {
        acc_val = lerp(acc_val, make_float4(1.0f, 0.0f, 0.0f, 0.f), 1.0f / static_cast<float>( frame + 1 ));
        }
        if (prd.depth > 1 && prd.visible && prd.radiance.x > 0.001f && prd.radiance.y > 0.001f && prd.radiance.z > 0.001f)
        {
        acc_val = lerp(acc_val, make_float4(0.7f, 0.5f, 0.0f, 0.f), 1.0f / static_cast<float>( frame + 1 ));
        }
        if (prd.depth > 2 && prd.visible && prd.radiance.x > 0.001f && prd.radiance.y > 0.001f && prd.radiance.z > 0.001f)
        {
        acc_val = lerp(acc_val, make_float4(0.0f, 1.0f, 0.0f, 0.f), 1.0f / static_cast<float>( frame + 1 ));
        }
        if (prd.depth > 3 && prd.visible && prd.radiance.x > 0.001f && prd.radiance.y > 0.001f && prd.radiance.z > 0.001f)
        {
            acc_val = lerp(acc_val, make_float4(0.0f, 0.0f, 1.0f, 0.f), 1.0f / static_cast<float>( frame + 1 ));
        }
#endif
        }else {
            acc_val = make_float4(prd.radiance, 0.f);
        }

        output_id_buffer[entry_point_id][launch_index] = make_color(make_float3(acc_val));
        output_id_accum_buffer[entry_point_id][launch_index] = acc_val;
        if (frame == 0)
        {
            output_id_intensity_buffer[entry_point_id][launch_index] = 0.0f;
        }
        else if(prd.visible && prd.radiance.x >0.001f && prd.radiance.y > 0.001f && prd.radiance.z > 0.001f){
            output_id_intensity_buffer[entry_point_id][launch_index] = prd.intensity;
        }
        // Change here for a gamma corrected RGB picture
//        output_id_buffer[entry_point_id][launch_index] = make_color(make_float3(powf(acc_val.x, 1.0f / 2.2f), powf(acc_val.y, 1.0f / 2.2f) , powf(acc_val.z, 1.0f / 2.2f)));
    }
}
// Entry point program
