#include "hip/hip_runtime.h"
#include "includes/app_config.h"
#include "includes/vertex_attributes.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optix_cuda.h>

rtBuffer<VertexAttributes> attributesBuffer;
rtBuffer<uint3>            indicesBuffer;

// Attributes.
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(optix::float3, tangent,   attribute tangent, );
rtDeclareVariable(optix::float3, tex_coord,  attribute tex_coord, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

// Intersection routine for indexed interleaved triangle data.
RT_PROGRAM void intersect(int primitiveIndex)
{
    const uint3 indices = indicesBuffer[primitiveIndex];

    VertexAttributes const& a0 = attributesBuffer[indices.x];
    VertexAttributes const& a1 = attributesBuffer[indices.y];
    VertexAttributes const& a2 = attributesBuffer[indices.z];

    const float3 v0 = a0.vertex;
    const float3 v1 = a1.vertex;
    const float3 v2 = a2.vertex;

    float3 n;
    float  t;
    float  beta;
    float  gamma;

    if (intersect_triangle(ray, v0, v1, v2, n, t, beta, gamma))
    {
        if (rtPotentialIntersection(t))
        {
            // Barycentric interpolation:
            const float alpha = 1.0f - beta - gamma;

            // Note: No normalization on the TBN attributes here for performance reasons.
            //       It's done after the transformation into world space anyway.
            geometric_normal      = n;
            tangent        = a0.tangent  * alpha + a1.tangent  * beta + a2.tangent  * gamma;
            shading_normal         = a0.normal   * alpha + a1.normal   * beta + a2.normal   * gamma;
            tex_coord       = a0.texcoord * alpha + a1.texcoord * beta + a2.texcoord * gamma;

            rtReportIntersection(0);
        }
    }
}
// Axis Aligned Bounding Box routine for indexed interleaved triangle data.
RT_PROGRAM void bounds(int primitiveIndex, float result[6])
{
    const uint3 indices = indicesBuffer[primitiveIndex];

    const float3 v0 = attributesBuffer[indices.x].vertex;
    const float3 v1 = attributesBuffer[indices.y].vertex;
    const float3 v2 = attributesBuffer[indices.z].vertex;

    const float area = optix::length(optix::cross(v1 - v0, v2 - v0));

    optix::Aabb *aabb = (optix::Aabb *) result;

    if (0.0f < area && !isinf(area))
    {
        aabb->m_min = fminf(fminf(v0, v1), v2);
        aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
    }
    else
    {
        aabb->invalidate();
    }
}
