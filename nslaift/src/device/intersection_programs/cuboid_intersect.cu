#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optix_cuda.h>

rtDeclareVariable(optix::Matrix4x4, Rt, , );
///< Maybe replace these later with Transformation matrix and x,y,z direction of the cubiod
rtDeclareVariable(float3, boxmin, , );
rtDeclareVariable(float3, boxmax, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

static __device__ float3 boxnormal(float t, float3 t0, float3 t1)
{
    float3 neg = make_float3(t==t0.x?1:0, t==t0.y?1:0, t==t0.z?1:0);
    float3 pos = make_float3(t==t1.x?1:0, t==t1.y?1:0, t==t1.z?1:0);
    return pos-neg;
}

RT_PROGRAM void intersect(int)
{
    float3 t0 = (boxmin - ray.origin)/ray.direction;
    float3 t1 = (boxmax - ray.origin)/ray.direction;
    float3 near = fminf(t0, t1);
    float3 far = fmaxf(t0, t1);
    float tmin = fmaxf( near );
    float tmax = fminf( far );

    if(tmin <= tmax) {
        bool check_second = true;
        if( rtPotentialIntersection( tmin ) ) {
            shading_normal = geometric_normal = boxnormal( tmin, t0, t1 );
            if(rtReportIntersection(0))
                check_second = false;
        }
        if(check_second) {
            if( rtPotentialIntersection( tmax ) ) {
                shading_normal = geometric_normal = boxnormal( tmax, t0, t1 );
                rtReportIntersection(0);
            }
        }
    }
}

RT_PROGRAM void bounds (int, float result[6])
{
    optix::Aabb* aabb = (optix::Aabb*)result;
    aabb->set(boxmin-2, boxmax+2);
}

