#include "hip/hip_runtime.h"

#include "includes/app_config.h"
#include "includes/light_definition.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_cuda.h>
#include "optix_math.h"

rtDeclareVariable(optix::Matrix4x4, Rt, ,);
rtDeclareVariable(optix::Matrix4x4, Rt_inv, ,);
rtDeclareVariable(optix::float3, color, ,);

RT_CALLABLE_PROGRAM void light(float3 const& point, const float2 sample, LightDefinition& lightDef)
{
    float3 light_position = make_float3(Rt.getCol(3));
    lightDef.distance = length(light_position - point);
    lightDef.wi = normalize(light_position - point); //light_definition direction from surface to light_definition
    lightDef.area = 1.0f ;
    lightDef.solid_angle = 4.0f * M_PIf* lightDef.area / (powf(fmaxf(lightDef.distance , DENOMINATOR_EPSILON ), 2.0f));

    float4 point_4f = make_float4(point);
    point_4f.w = 1.0f;
    lightDef.emission = color;
}
